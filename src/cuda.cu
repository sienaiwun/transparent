#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

texture<float4, 2, hipReadModeElementType> cudaEdgeTex;
texture<float4, 2, hipReadModeElementType> cudaOccuderTex;
texture<float4, 2, hipReadModeElementType> cudaColorTex;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

__device__ bool isVolume(float2 uv, int *state)
{
	if (uv.x > 230 && uv.x < 300)
		return true;
	float4 value = tex2D(cudaOccuderTex, uv.x, uv.y);
	return value.x > 0.5;
}
typedef enum {
	isVolumn,
	notVolumn,
}pixelEnum;
class ListNote
{
public:

	unsigned int beginIndex, endIndex, nextPt;
};

uint4 *cuda_PBO_Buffer;

__device__ uint4* d_cudaPboBuffer;
float4 *cuda_TexturePbo_buffer;
__device__ float4* d_cudaTexture;
__device__ int imageWidth, imageHeight, d_outTextureWidth, d_outTextureHeigh;
__device__ int d_index;
__device__ ListNote* d_listBuffer;
__device__ int d_atomic;

class List
{
	
};
__global__ void countRowKernel(int kernelWidth, int kernelHeight)
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if ( y > kernelHeight)
		return;
	//if (y != 599)
	//	return;
	int arrayNum = y;
	int accumNum = 0;
	int state = 0;
	pixelEnum etype = notVolumn;
	unsigned int* nextPtr = &d_cudaPboBuffer[arrayNum].x;
	int listIndex;
	for (int x = 0; x < imageWidth;x++)
	{
		float2 currentUv = make_float2(x + 0.5, y + 0.5);
		if (isVolume(currentUv, &state) && etype == notVolumn)
		{
			//printf("insert :%d\n", x);
			listIndex = atomicAdd(&d_atomic, 1);
			atomicExch(nextPtr, listIndex);// write listIndex to next slot
			d_listBuffer[listIndex].beginIndex = x;
			d_listBuffer[listIndex].endIndex = x + 1;
			d_listBuffer[listIndex].nextPt = 0;
			nextPtr = (unsigned int *)(&(d_listBuffer[listIndex].nextPt));
			etype = isVolumn;
			
		}
		else if (isVolume(currentUv, &state) && etype == isVolumn)
		{
			
			
		}
		else if (!isVolume(currentUv, &state) && etype == isVolumn)
		{
			//printf("end :%d\n", x);

			d_listBuffer[listIndex].endIndex = x;
			etype = notVolumn;
		}

	}
}
__global__ void renderToTexutre(int kernelWidth, int kernelHeight)
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (y > kernelHeight)
		return;
	
}
ListNote *device_data = NULL;
int atomBuffer = 0;
#ifdef DEBUG
	ListNote *host_data = NULL;
#endif
extern void cudaInit(int height, int k)
{
	checkCudaErrors(hipMalloc(&device_data, height*k*sizeof(ListNote)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_listBuffer), &device_data,sizeof(ListNote*)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(device_data,0, height*k*sizeof(ListNote)));  
#ifdef DEBUG
	checkCudaErrors(hipHostMalloc(&host_data, height*k*sizeof(ListNote)));
#endif
	//host_data = (ListNote*)malloc(height*k*sizeof(ListNote));
	//memset(host_data, 0, height*k*sizeof(ListNote));
	//checkCudaErrors(hipMemcpy((void *)device_data, (void *)host_data, height * k * sizeof(ListNote), hipMemcpyDeviceToHost));

	
}
extern "C" void countRow(int width, int height)
{

	hipEvent_t begin_t, end_t;
	checkCudaErrors(hipEventCreate(&begin_t));
	checkCudaErrors(hipEventCreate(&end_t));

	hipEventRecord(begin_t, 0);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(cuda_PBO_Buffer, 0, height*sizeof(uint4)));

	dim3 blockSize(1, 16, 1);
	dim3 gridSize(1, height / blockSize.y, 1);
	countRowKernel << <gridSize, blockSize >> >(1, height);
	hipEventRecord(end_t, 0);
	hipEventSynchronize(end_t);
	float costtime;
	checkCudaErrors(hipEventElapsedTime(&costtime, begin_t, end_t));

	checkCudaErrors(hipEventDestroy(begin_t));
	checkCudaErrors(hipEventDestroy(end_t));

#ifdef DEBUG
	int arraySize = 0;
	checkCudaErrors(hipMemcpy((void *)host_data, (void *)device_data, height*10*sizeof(ListNote), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(&arraySize, HIP_SYMBOL(d_atomic), sizeof(int)));

	for (int i = 0; i < arraySize; i++)
	{
		printf("b:%d,n:%d,next:%d\n", host_data[i].beginIndex, host_data[i].endIndex, host_data[i].nextPt);
	}
#endif

}

extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint();
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray, *pCudaTex, 0, 0));
	int w = pResouce->getWidth();
	int h = pResouce->getHeight();
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(imageWidth), &w, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(imageHeight), &h, sizeof(int)));
	if (occluderbuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaOccuderTex, tmpcudaArray, channelDesc));
		cudaOccuderTex.filterMode = hipFilterModeLinear;
	}
	else if (edgebuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaEdgeTex, tmpcudaArray, channelDesc));
		cudaEdgeTex.filterMode = hipFilterModePoint;
	}
	else if (color_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaColorTex, tmpcudaArray, channelDesc));
		cudaColorTex.filterMode = hipFilterModePoint;
	}

}
extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint();
	int w = pResource->getWidth();
	int h = pResource->getHeight();
	if (unit_4 == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer, sizeof(uint4*)));
	}
	else if (float4_t == pResource->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureWidth),  &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureHeigh), &h, sizeof(int)));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_TexturePbo_buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTexture), &cuda_TexturePbo_buffer, sizeof(float4*)));
	}
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

texture<float4, 2, hipReadModeElementType> cudaEdgeTex;
texture<float4, 2, hipReadModeElementType> cudaOccuderTex;
texture<float4, 2, hipReadModeElementType> cudaColorTex;
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

typedef enum {
	isVolumn,
	notVolumn,
}pixelEnum;
class ListNote
{
public:

	unsigned int nextPt,beginIndex, endIndex;
};

uint3 *cuda_PBO_Buffer;

__device__ uint3* d_cudaPboBuffer;
float4 *cuda_TexturePbo_buffer;
__device__ float4* d_cudaTexture;
__device__ int imageWidth, imageHeight, d_outTextureWidth, d_outTextureHeigh;
__device__ int d_index;
__device__ ListNote* d_listBuffer;
__device__ int d_atomic;

class List
{
	
};

__device__ bool isVolume(float2 uv, int *state)
{
	//if (uv.x > 230 && uv.x < 300)
	//	return true;
	float4 value = tex2D(cudaOccuderTex, uv.x, uv.y);
	return value.x > 0.5;
}
__device__ float2 toUv(int x, int y)
{
	return make_float2(x + 0.5, y + 0.5);
}
__global__ void countRowKernel(int kernelWidth, int kernelHeight)
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if ( y > kernelHeight)
		return;
	//if (y != 599)
	//	return;
	int arrayNum = y;
	int accumNum = 0;
	int state = 0;
	pixelEnum etype = notVolumn;
	unsigned int* nextPtr = &d_cudaPboBuffer[arrayNum].x;
	int listIndex;
	for (int x = 0; x < imageWidth;x++)
	{
		float2 currentUv = toUv(x, y);
		if (isVolume(currentUv, &state) && etype == notVolumn)
		{
			//printf("insert :%d\n", x);
			listIndex = atomicAdd(&d_atomic, 1);
			atomicExch(nextPtr, listIndex);// write listIndex to next slot
			d_listBuffer[listIndex].beginIndex = x;
			d_listBuffer[listIndex].endIndex = x + 1;
			d_listBuffer[listIndex].nextPt = 0;
			nextPtr = (unsigned int *)(&(d_listBuffer[listIndex].nextPt));
			etype = isVolumn;
			
		}
		else if (isVolume(currentUv, &state) && etype == isVolumn)
		{
			
			
		}
		else if (!isVolume(currentUv, &state) && etype == isVolumn)
		{
			//printf("end :%d\n", x);

			d_listBuffer[listIndex].endIndex = x;
			etype = notVolumn;
		}

	}
}
__device__ void FillVolumn(int beginX, int endX, int y)
{
	int top = min(endX, d_outTextureWidth);
	//printf("volumn begin:%d,end:%d,top:%d\n",beginX,endX,top);
	for (int x = beginX; x < top; x++)
	{
		int index = y*d_outTextureWidth + x;
		d_cudaTexture[index] =make_float4(1,0,0,1);
		//printf("fillPixel(%d,%d),index%d  (%f,%f)\t", x, y, index, uvx, beginUv.y);

	}
}
__device__ void FillSpan(int beginX, int endX, int y,float2 beginUv,float2 endUv)
{
	int top = min(endX, d_outTextureWidth);
	//printf("tx: begin:%d,end:%d,top:%d,(%f,%f)\n",beginX,endX,top,beginUv.x,endUv.x);
	for (int x = beginX; x < top; x++)
	{
		int index = y*d_outTextureWidth+x;
		float uvx = beginUv.x + (endUv.x - beginUv.x) / (top - 1)*x;
		d_cudaTexture[index] = tex2D(cudaColorTex, uvx, beginUv.y);
		//printf("fillPixel(%d,%d),index%d  (%f,%f)\t", x, y, index, uvx, beginUv.y);

	}
}
__global__ void renderToTexutre(int kernelWidth, int kernelHeight)
{
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	if (y > kernelHeight)
		return;
	int listIndex = y;
	int rowLength = imageWidth;
	ListNote currentNote =* ((ListNote*)&d_cudaPboBuffer[listIndex]);
	//if (y != 599)
	//	return;
	int texEnd =0;
	int texBegin = 0;
	int fillBegin = 0;
	int fillEnd = 0;
	int acuumPixel =0,span =0;
	//("begin:%d,end%d,index:%d\n", currentNote.beginIndex, currentNote.endIndex, currentNote.nextPt);
	//printf("init:%d\n", d_cudaPboBuffer[listIndex].x);
	while (currentNote.nextPt != 0)
	{
		currentNote = d_listBuffer[currentNote.nextPt];
		texEnd = currentNote.endIndex;
		span = currentNote.endIndex - currentNote.beginIndex;
		fillBegin = texBegin + acuumPixel;
		fillEnd = texEnd + acuumPixel;
		FillSpan(fillBegin, fillEnd, y, toUv(texBegin, y), toUv(texEnd, y));
		FillVolumn(fillEnd, fillEnd+span, y);
		
		acuumPixel += span;
		texBegin = currentNote.endIndex;
		
	}
	fillBegin = texBegin + acuumPixel;
	//printf("final:(%d,%d) u(%f,%f)\n", fillBegin, imageWidth + span, toUv(texBegin, y).x, toUv(imageWidth - 1, y).x);

	FillSpan(fillBegin, imageWidth + span, y, toUv(texBegin, y), toUv(imageWidth - 1, y));

	
}
ListNote *device_data = NULL;
int atomBuffer = 0;
#ifdef DEBUG
	ListNote *host_data = NULL;
#endif
extern void cudaInit(int height, int width,int k,int rowLarger)
{
	checkCudaErrors(hipMalloc(&device_data, height*k*sizeof(ListNote)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_listBuffer), &device_data,sizeof(ListNote*)));
	
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(device_data,0, height*k*sizeof(ListNote)));  
	//checkCudaErrors(hipMemset(cuda_TexturePbo_buffer, 0, width* height*rowLarger*sizeof(float4)));
#ifdef DEBUG
	checkCudaErrors(hipHostMalloc(&host_data, height*k*sizeof(ListNote)));
#endif
	//host_data = (ListNote*)malloc(height*k*sizeof(ListNote));
	//memset(host_data, 0, height*k*sizeof(ListNote));
	//checkCudaErrors(hipMemcpy((void *)device_data, (void *)host_data, height * k * sizeof(ListNote), hipMemcpyDeviceToHost));

	
}
extern "C" void countRow(int width, int height)
{

	hipEvent_t begin_t, end_t;
	checkCudaErrors(hipEventCreate(&begin_t));
	checkCudaErrors(hipEventCreate(&end_t));

	hipEventRecord(begin_t, 0);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_atomic), &atomBuffer, sizeof(int)));
	checkCudaErrors(hipMemset(cuda_PBO_Buffer, 0, height*sizeof(uint3)));
	checkCudaErrors(hipMemset(cuda_TexturePbo_buffer, 0, ROWLARGER*width*height*sizeof(float4)));

	dim3 blockSize(1, 16, 1);
	dim3 gridSize(1, height / blockSize.y, 1);
	countRowKernel << <gridSize, blockSize >> >(1, height);
	hipEventRecord(end_t, 0);
	hipEventSynchronize(end_t);
	float costtime;
	checkCudaErrors(hipEventElapsedTime(&costtime, begin_t, end_t));

	renderToTexutre << <gridSize, blockSize >> >(1, height);

	checkCudaErrors(hipEventDestroy(begin_t));
	checkCudaErrors(hipEventDestroy(end_t));

#ifdef DEBUG
	int arraySize = 0;
	checkCudaErrors(hipMemcpy((void *)host_data, (void *)device_data, height*10*sizeof(ListNote), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpyFromSymbol(&arraySize, HIP_SYMBOL(d_atomic), sizeof(int)));

	for (int i = 0; i < arraySize; i++)
	{
		printf("b:%d,n:%d,next:%d\n", host_data[i].beginIndex, host_data[i].endIndex, host_data[i].nextPt);
	}
#endif

}

extern "C"  void cudaRelateTex(CudaTexResourse * pResouce)
{

	hipArray *tmpcudaArray;
	hipGraphicsResource ** pCudaTex = pResouce->getResPoint();
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&tmpcudaArray, *pCudaTex, 0, 0));
	int w = pResouce->getWidth();
	int h = pResouce->getHeight();
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(imageWidth), &w, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(imageHeight), &h, sizeof(int)));
	if (occluderbuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaOccuderTex, tmpcudaArray, channelDesc));
		cudaOccuderTex.filterMode = hipFilterModeLinear;
	}
	else if (edgebuffer_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaEdgeTex, tmpcudaArray, channelDesc));
		cudaEdgeTex.filterMode = hipFilterModePoint;
	}
	else if (color_t == pResouce->getType())
	{
		checkCudaErrors(hipBindTextureToArray(cudaColorTex, tmpcudaArray, channelDesc));
		cudaColorTex.filterMode = hipFilterModePoint;
	}

}
extern "C" void cudaRelateArray(CudaPboResource * pResource)
{
	size_t numBytes;
	hipGraphicsResource ** pCudaTex = pResource->getResPoint();
	int w = pResource->getWidth();
	int h = pResource->getHeight();
	if (unit_3 == pResource->getType())
	{
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_PBO_Buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaPboBuffer), &cuda_PBO_Buffer, sizeof(uint3*)));
	}
	else if (float4_t == pResource->getType())
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureWidth),  &w, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_outTextureHeigh), &h, sizeof(int)));

		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&cuda_TexturePbo_buffer, &numBytes, *pCudaTex));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_cudaTexture), &cuda_TexturePbo_buffer, sizeof(float4*)));
	}
}
